#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
#include <iostream>
#include <chrono>
#include <vector>
#include <algorithm>
#include <execution>
#include <sstream>
#include <string>
#include "time.h"
#include <filesystem>
#include <cmath>
using namespace cv;
using namespace std::filesystem;
void logCurrentTime()
{
	// 获取当前时间点
	auto now = std::chrono::system_clock::now();
	// 转换为time_t格式，以便使用ctime库
	std::time_t now_time_t = std::chrono::system_clock::to_time_t(now);
	// 将time_t转换为本地时间
	std::tm now_tm;
	localtime_s(&now_tm, &now_time_t);
	std::cout << "当前时间是: " << std::put_time(&now_tm, "%Y-%m-%d %X") << "\n";
}
int scale = 4096;
Mat dataSet;
Mat fracImage;
Vec3b black = Vec3b(0, 0, 0);
Vec3b white = Vec3b(255, 255, 255);
/*void processorTemplate(int i, int j, Vec3b& color, Mat img)
{

}*/
struct Vec3uchar
{
	uchar x;
	uchar y;
	uchar z;
	__device__ Vec3uchar(uchar value)
	{
		this->x = this->y = this->z = value;
	}
	__device__ Vec3uchar(uchar x, uchar y, uchar z)
	{
		this->x = x;
		this->y = y;
		this->z = z;
	}
	__device__ Vec3uchar operator+(const Vec3uchar& rhs) const {
		return Vec3uchar(x + rhs.x, y + rhs.y, z + rhs.z);
	}
	__device__ Vec3uchar operator-(const Vec3uchar& rhs) const {
		return Vec3uchar(x - rhs.x, y - rhs.y, z - rhs.z);
	}
	__device__ Vec3uchar operator*(const Vec3uchar& rhs) const {
		return Vec3uchar(x * rhs.x, y * rhs.y, z * rhs.z);
	}
	__device__ Vec3uchar operator/(const Vec3uchar& rhs) const {
		return Vec3uchar(x / rhs.x, y / rhs.y, z / rhs.z);
	}
	friend __device__  Vec3uchar operator*(const Vec3uchar& p, uchar scalar) {
		return Vec3uchar(p.x * scalar, p.y * scalar, p.z * scalar);
	}
	friend __device__  Vec3uchar operator/(const Vec3uchar& p, uchar scalar) {
		return Vec3uchar(p.x / scalar, p.y / scalar, p.z / scalar);
	}
	__device__ Vec3uchar swapElem()
	{
		return Vec3uchar(this->y, this->z, this->x);
	}
};
void processorBlackWhite(Vec3b& color)
{
	color = (color[0] + color[1] + color[2]) > 255.0 ? white : black;
	//auto orig = color;
	//double value = color[0] / 255.0 + color[1] / 255.0 + color[2] / 255.0;
	//uchar c = value > 0.33 ? 255 : 0;
	//color = Vec3b(c, c, c);
}
template<typename T>
__host__ __device__ T clamp(T value, T min, T max)
{
	if (value > max)return max;
	if (value < min)return min;
	return value;
}
int lengthSquared(Vec2i vec)
{
	return vec[0] * vec[0] + vec[1] * vec[1];
}
Vec3b IntToColor(int p)
{
	return Vec3b((uchar)p, (uchar)(p >> 8), (uchar)(p >> 16));

}
void processorESSEDT(Mat img)
{
	//初始化

	int width = img.cols;
	int height = img.rows;
	Vec2i** deltaS = new Vec2i * [height];
	Vec2i* data = new Vec2i[height * width];
	for (int i = 0; i < height; ++i) {
		deltaS[i] = &data[i * width];
	}
	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++)
		{

			deltaS[i][j] = (img.at<Vec3b>(i, j)[0] > 20) ? Vec2i(0, 0) : Vec2i(width, height);
		}


	// 第一个像素(左上)
	{
		bool flag = true;
		int counter = 0;//计数器，表示对当前像素查找的次数
		Vec2i unit = Vec2i(0, 0);
		float dist = 0;
		while (flag)
		{
			int x = counter % scale;
			int y = counter / scale * 2;
			Vec3b xData = dataSet.at<Vec3b>(y, x);//从数据图中获得xy偏移量
			Vec3b yData = dataSet.at<Vec3b>(y + 1, x);
			unit = Vec2i((xData[2] * 256 + xData[1]) * 256 + xData[0], (yData[2] * 256 + yData[1]) * 256 + yData[0]);//生成偏移向量
			for (int n = 0; n < 2; n++)
			{
				Vec2i _unit = unit;
				//以下三行对应三个对称操作，由一个偏移向量生成等模长的三个
				if (n > 0) _unit = Vec2i(_unit[1], _unit[0]);

				//查询格点，如果是白色像素就停止(只有两个状态，所以我直接x>0了
				if (img.at<Vec3b>(clamp(_unit[1], 0, height - 1), clamp(_unit[0], 0, width - 1))[0] > 0)
				{
					flag = false;//停止当前像素的查找
					dist = lengthSquared(unit);//记录该像素到最近白色像素的距离的平方
					unit = _unit;
					break;
				}
			}
			counter++;//查询次数自增
		}
		deltaS[0][0] = unit;
	}
	// 上到下扫描
	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++)
		{
			Vec2i& cur = deltaS[i][j];
			if (cur[0] == 0 && cur[1] == 0) continue;
			int length = lengthSquared(cur);
			if (j != 0)
			{
				Vec2i tar = deltaS[i][j - 1] + Vec2i(-1, 0);
				int lengthTar = lengthSquared(tar);
				if (lengthTar < length)
				{
					cur = tar;
					length = lengthTar;
				}

			}
			if (i != 0)
			{
				Vec2i tar = deltaS[i - 1][j] + Vec2i(0, -1);
				int lengthTar = lengthSquared(tar);
				if (lengthTar < length)
				{
					cur = tar;
					length = lengthTar;

				}
				if (j != 0)
				{
					tar = deltaS[i - 1][j - 1] + Vec2i(-1, -1);
					lengthTar = lengthSquared(tar);
					if (lengthTar < length)
					{
						cur = tar;
						length = lengthTar;
					}
				}
				if (j != width - 1)
				{
					tar = deltaS[i - 1][j + 1] + Vec2i(1, -1);
					lengthTar = lengthSquared(tar);
					if (lengthTar < length)
					{
						cur = tar;
						length = lengthTar;
					}
				}
			}
		}
	// 第一个像素(右下)
	{
		bool flag = true;
		int counter = 0;//计数器，表示对当前像素查找的次数
		Vec2i unit = Vec2i(0, 0);
		float dist = 0;
		while (flag)
		{
			int x = counter % scale;
			int y = counter / scale * 2;
			Vec3b xData = dataSet.at<Vec3b>(y, x);//从数据图中获得xy偏移量
			Vec3b yData = dataSet.at<Vec3b>(y + 1, x);
			unit = Vec2i((xData[2] * 256 + xData[1]) * 256 + xData[0], (yData[2] * 256 + yData[1]) * 256 + yData[0]);//生成偏移向量

			for (int n = 0; n < 2; n++)
			{
				Vec2i _unit = unit;
				//以下三行对应三个对称操作，由一个偏移向量生成等模长的三个
				if (n > 0) _unit = Vec2i(_unit[1], _unit[0]);
				_unit *= -1;
				_unit += Vec2i(width - 1, height - 1);

				//查询格点，如果是白色像素就停止(只有两个状态，所以我直接x>0了
				if (img.at<Vec3b>(clamp(_unit[1], 0, height - 1), clamp(_unit[0], 0, width - 1))[0] > 0)
				{
					flag = false;//停止当前像素的查找
					dist = lengthSquared(unit);//记录该像素到最近白色像素的距离的平方
					unit = _unit;
					break;
				}
			}
			counter++;//查询次数自增
		}
		Vec2i tar = unit;// -new Vector2(width - 1, height - 1);
		if (lengthSquared(tar) < lengthSquared(deltaS[height - 1][width - 1]))
			deltaS[height - 1][width - 1] = tar;
	}
	// 下到上扫描
	for (int i = height - 1; i >= 0; i--)
		for (int j = width - 1; j >= 0; j--)
		{
			Vec2i& cur = deltaS[i][j];
			if (cur[0] == 0 && cur[1] == 0) continue;
			int length = lengthSquared(cur);

			if (j != width - 1)
			{
				Vec2i tar = deltaS[i][j + 1] + Vec2i(1, 0);
				int lengthTar = lengthSquared(tar);
				if (lengthTar < length)
				{
					cur = tar;
					length = lengthTar;
				}

			}
			if (i != height - 1)
			{
				Vec2i tar = deltaS[i + 1][j] + Vec2i(0, 1);
				int lengthTar = lengthSquared(tar);
				if (lengthTar < length)
				{
					cur = tar;
					length = lengthTar;
				}
				if (j != 0)
				{
					tar = deltaS[i + 1][j - 1] + Vec2i(-1, 1);
					lengthTar = lengthSquared(tar);
					if (lengthTar < length)
					{
						cur = tar;
						length = lengthTar;
					}
				}
				if (j != width - 1)
				{
					tar = deltaS[i + 1][j + 1] + Vec2i(1, 1);
					lengthTar = lengthSquared(tar);
					if (lengthTar < length)
					{
						cur = tar;
						length = lengthTar;
					}
				}
			}
		}
	for (int j = 0; j < width; j++)
		for (int i = 0; i < height; i++)
		{
			img.at<Vec3b>(i, j) = IntToColor(lengthSquared(deltaS[i][j]));//用像素来记录距离信息
		}

	delete[] data;  // 释放整个内存块
	delete[] deltaS;  // 释放行指针数组
}
void processorFractal(double angle, Vec3b& color)
{
	int distSqr = color[0] + 255 * (color[1] + 255 * color[2]);//把像素信息转距离信息
	Vec2d orig = Vec2d(-32 / 9.0, -2.0);//缩放中心
	//double angle = atan2(i - height * .5, j - width * .5) * 4;
	Vec2d vec = Vec2d(cos(angle), sin(angle)) * 0.5;
	angle *= 2;
	vec -= Vec2d(cos(angle), sin(angle)) * 0.25;
	vec *= 0.95 + clamp(sqrt(distSqr) / 8000.0, 0.0, 1000.0) * 64.0;
	vec -= orig;
	vec *= 270;
	color = fracImage.at<Vec3b>(clamp((int)vec[1], 0, 1079), clamp((int)vec[0], 0, 1919));
}
class PixelOperation_BlackWhite : public cv::ParallelLoopBody {
public:
	PixelOperation_BlackWhite(Mat& _img) : img(_img) {}

	void operator()(const cv::Range& range) const {
		for (int i = range.start; i < range.end; i++) {
			for (int j = 0; j < img.cols; j++) {
				processorBlackWhite(img.at<Vec3b>(i, j));
			}
			//std::cout << "Processed row " << i << std::endl;
		}
	}

private:
	Mat& img;
};
class PixelOperation_Fractal : public cv::ParallelLoopBody {
public:
	PixelOperation_Fractal(Mat& _img) : img(_img) {}

	void operator()(const cv::Range& range) const {
		int width = img.cols;
		int height = img.rows;
		for (int i = range.start; i < range.end; i++) {
			for (int j = 0; j < width; j++) {
				processorFractal(atan2(i - height * .5, j - width * .5) * 4, img.at<Vec3b>(i, j));
			}
			//std::cout << "Processed row " << i << std::endl;
		}
	}

private:
	Mat& img;
};


__global__ void kernel_BlackWhite(Vec3uchar* data, Vec3uchar* dest, int n, uchar standard)
{
	int offset = n * blockIdx.x;
	for (int k = threadIdx.x; k < n; k += blockDim.x)
	{
		auto vec = data[k + offset] / 3;
		dest[k + offset] = Vec3uchar((vec.x + vec.y + vec.z) > standard ? 255 : 0);
	}
}
void processImage_BlackWhite(Mat img, Mat dest, int standard)
{
	int width = img.cols;
	int height = img.rows;
	Vec3uchar* data;
	Vec3uchar* data_dest;

	//选择使用哪个GPU运行
	//hipSetDevice(0);
	auto size = height * width * 3 * sizeof(uchar);
	hipMalloc((void**)&data, size);
	hipMalloc((void**)&data_dest, size);
	hipMemcpy(data, img.ptr(0), size, hipMemcpyHostToDevice);

	kernel_BlackWhite << <height, 1024 >> > (data, data_dest, width, (uchar)standard);
	hipDeviceSynchronize();
	hipMemcpy(dest.ptr(0), data_dest, size, hipMemcpyDeviceToHost);
	hipFree(data);
	hipFree(data_dest);

}

__global__ void kernel_Fractal(Vec3uchar* data, Vec3uchar* dest, Vec3uchar* fracImage, int n)
{
	int offset = n * blockIdx.x;
	for (int k = threadIdx.x; k < n; k += blockDim.x)
	{
		auto vec = data[k + offset];
		float angle = atan2(k - gridDim.x * .5, blockIdx.x - n * .5) * 4;
		int distSqr = vec.x + 255 * (vec.y + 255 * vec.z);//把像素信息转距离信息
		float coordX = cos(angle) * .5f;
		float coordY = sin(angle) * .5f;
		angle *= 2;
		coordX -= cos(angle) * .25f;
		coordY -= sin(angle) * .25f;
		angle = 0.95f + clamp(sqrtf(distSqr) / 8000.0f, 0.0f, 1000.0f) * 64.0f;
		coordX *= angle;
		coordY *= angle;
		coordX += 32 / 9.0f;
		coordY += 2.0f;
		coordX *= 270;
		coordY *= 270;
		dest[k + offset] = fracImage[clamp((int)coordX, 0, 1919) + 1920 * clamp((int)coordY, 0, 1079)];
	}
}


void processImage_Fractal(Mat img, Mat dest, Vec3uchar* fracImage)
{
	int width = img.cols;
	int height = img.rows;
	Vec3uchar* data;
	Vec3uchar* data_dest;

	//选择使用哪个GPU运行
	//hipSetDevice(0);
	auto size = height * width * 3 * sizeof(uchar);
	hipMalloc((void**)&data, size);
	hipMalloc((void**)&data_dest, size);
	hipMemcpy(data, img.ptr(0), size, hipMemcpyHostToDevice);
	kernel_Fractal << <height, 1024 >> > (data, data_dest, fracImage, width);
	hipDeviceSynchronize();
	hipMemcpy(dest.ptr(0), data_dest, size, hipMemcpyDeviceToHost);
	hipFree(data);
	hipFree(data_dest);
}

int main(int argc, char* argv[])
{
	if (argc == 1)
	{
		std::cout << "请尝试拖入一些文件来给exe处理吧\n";
		std::cin.get();
		return 0;
	}
	path p = path(argv[0]).parent_path();
	dataSet = imread((p / "dataSet4096.png").string(), IMREAD_COLOR);
	fracImage = imread((p / "WallPaper_FractalMandelbort.png").string(), IMREAD_COLOR);

	std::cout << "请输入相应数字来执行相应功能\n";
	std::cout << "0 将画面进行裁切(3840x2160→2880x2160)\n";
	std::cout << "1 将画面阈值处理，亮度高于0.33的改成白色，否则是黑色，亮度∈[0,1]\n";
	std::cout << "2 将 黑白 画面进行距离场处理，计算每个像素到最近的白色像素的距离的平方并且以颜色形式存储\n";
	std::cout << "3 将 距离场 图进行分形映射处理\n";
	std::cout << "4 除了裁切以外的一条龙服务，适用于大多数图(图片亮度太低会在1处理成纯黑然后2炸掉)\n";
	int index;
	std::cin >> index;
	if (index < 0 || index > 4)
	{
		std::cout << "不认识的数字呢\n";
		std::cin.get();
		std::cin.get();
		return 0;
	}
	int standard_BlackWhite = 0;
	Vec3uchar* frac;
	if (index == 1 || index == 4)
	{
		std::cout << "请输入黑白阈值(0-255)\n";
		std::cin >> standard_BlackWhite;
	}
	if (index == 3 || index == 4)
	{
		auto size = 1920 * 1080 * 3 * sizeof(uchar);
		hipMalloc((void**)&frac, size);
		hipMemcpy(frac, fracImage.ptr(0), size, hipMemcpyHostToDevice);
	}
	std::cout << "开始，";
	logCurrentTime();
	std::cout << "请耐心等待处理\n";
	//std::cout << "请耐心等待处理，多于1000张时每处理100张会输出一次进度，否则多于10张时每10张输出一次进度\n";
	int counter = 0;
	std::vector<char*> data;

	char** begin = argv + 1;
	char** end = argv + argc;
	std::copy(begin, end, std::back_inserter(data));

	auto processor = [frac, standard_BlackWhite, index, argc, &counter](char* charpath)
		{
			/*std::ostringstream ss;
			ss << path << "," << counter++ << "\n";
			std::string merged = ss.str();
			std::cout << merged;*/
			counter++;
			int c = counter;
			Mat img = imread(charpath, IMREAD_COLOR);
			switch (index)
			{
			case 0:
				std::cout << "我懒得做这个裁切了，这个是因为烂苹果是4：3而那个4k 60帧的是16：9我需要裁切一下才存在的\n";
				std::cin.get();
				std::cin.get();
				return 0;
			case 1:
				processImage_BlackWhite(img, img, standard_BlackWhite);
				break;
			case 2:
				processorESSEDT(img);
				break;
			case 3:
				processImage_Fractal(img, img, frac);
				break;
			case 4:
				processImage_BlackWhite(img, img, standard_BlackWhite);
				processorESSEDT(img);
				processImage_Fractal(img, img, frac);
				break;
			default:
				break;
			}
			path curpath = path(charpath);
			path sourceDir = curpath.parent_path();
			path resultDir = sourceDir / "Result_Cuda";
			if (!exists(resultDir)) {
				create_directories(resultDir);
			}
			//if (argc > 10 && c % (argc > 1000 ? 100 : 10) == 0)
			//{
			//	std::ostringstream ss;
			//	ss << (c * 100.0 / argc) << "%\n";
			//	std::string merged = ss.str();
			//	std::cout << merged;
			//}
			imwrite((resultDir / curpath.filename()).string(), img);

			//std::ostringstream _ss;
			//_ss << charpath << "_result.png";
			//std::string _merged = _ss.str();
			//imwrite(_merged, img);
		};
	std::vector<std::thread> threads;


	auto start = std::chrono::high_resolution_clock::now();
	for (int i = 0; i < data.size(); ++i) {
		threads.emplace_back(processor, data[i]);
	}
	// 等待所有线程完成
	for (auto& thread : threads) {
		thread.join();
	}
	auto stop = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double, std::milli> duration = stop - start;
	std::cout << "Function took " << duration.count() << " milliseconds." << std::endl;


	std::cout << "处理成功，已处理" << counter << "个文件，请查看它们自己目录下的Result_Cuda文件夹\n";
	std::cout << "结束，";
	logCurrentTime();
	if (index == 3 || index == 4)
	{
		hipFree(frac);
	}
	std::cout << "输入任意按键退出\n";
	std::cin.get();
	std::cin.get();


}